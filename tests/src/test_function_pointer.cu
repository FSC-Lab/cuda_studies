#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <random>

using BinaryOperation = void (*)(uint, double const*, double const*, double*);

#define MAKE_BINARY_OP(name, op)                                       \
  __device__ void name(uint len, double const* lhs, double const* rhs, \
                       double* res) {                                  \
    for (uint i = 0; i < len; ++i) {                                   \
      res[i] = lhs[i] op rhs[i];                                       \
    }                                                                  \
  }

MAKE_BINARY_OP(Add, +);
MAKE_BINARY_OP(Sub, -);
MAKE_BINARY_OP(Mul, *);
MAKE_BINARY_OP(Div, /);
// Required for functional pointer argument in kernel function
// Static pointers to device functions
__device__ BinaryOperation p_add = Add;
__device__ BinaryOperation p_sub = Sub;
__device__ BinaryOperation p_mul = Mul;
__device__ BinaryOperation p_div = Div;

__global__ void BinaryOperationKernel(BinaryOperation op, uint num_samples,
                                      uint len_x, double* lhs, double* rhs,
                                      double* res) {
  uint i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= num_samples) {
    return;
  }
  double* ith_lhs = lhs + i * len_x;
  double* ith_rhs = rhs + i * len_x;
  double* ith_res = res + i * len_x;
  (*op)(len_x, ith_lhs, ith_rhs, ith_res);
}

thrust::host_vector<double> InvokeBinaryOperation(
    const BinaryOperation& op, const thrust::host_vector<double>& lhs,
    uint num_samples, const thrust::host_vector<double>& rhs) {
  BinaryOperation p_op;
  hipMemcpyFromSymbol(&p_op, HIP_SYMBOL(op), sizeof(BinaryOperation));

  thrust::device_vector<double> d_lhs(lhs.cbegin(), lhs.cend());
  thrust::device_vector<double> d_rhs(rhs.cbegin(), rhs.cend());

  thrust::device_vector<double> d_res(lhs.size());
  const uint threads_per_block = 16;
  const uint num_blocks =
      (num_samples + threads_per_block - 1) / threads_per_block;
  const uint len_x = lhs.size() / num_samples;
  BinaryOperationKernel<<<threads_per_block, num_blocks>>>(
      p_op, num_samples, len_x, d_lhs.data().get(), d_rhs.data().get(),
      d_res.data().get());
  hipDeviceSynchronize();

  return {d_res.cbegin(), d_res.cend()};
}

struct TestFunctionPointer : public testing::Test {
  uint num_samples = 100;
  uint len_x = 4;

  std::random_device dev;
  std::mt19937 rng{dev()};
  std::uniform_real_distribution<> dist{10.0, 100.0};
  std::function<double()> generate_fn = std::bind(dist, std::ref(rng));

  thrust::host_vector<double> lhs;
  thrust::host_vector<double> rhs;
  thrust::host_vector<double> res;

  TestFunctionPointer()
      : ::testing::Test(), lhs(num_samples * len_x), rhs(num_samples * len_x) {
    thrust::generate(lhs.begin(), lhs.end(), generate_fn);
    thrust::generate(rhs.begin(), rhs.end(), generate_fn);
  }
};

TEST_F(TestFunctionPointer, testAddByFunctionPointer) {
  res = InvokeBinaryOperation(p_add, lhs, num_samples, rhs);

  for (int i = 0; i < num_samples * len_x; ++i) {
    ASSERT_NEAR(res[i], lhs[i] + rhs[i], 1e-5);
  }
}

TEST_F(TestFunctionPointer, testSubByFunctionPointer) {
  res = InvokeBinaryOperation(p_sub, lhs, num_samples, rhs);

  for (int i = 0; i < num_samples * len_x; ++i) {
    ASSERT_NEAR(res[i], lhs[i] - rhs[i], 1e-5);
  }
}

TEST_F(TestFunctionPointer, testMulByFunctionPointer) {
  res = InvokeBinaryOperation(p_mul, lhs, num_samples, rhs);

  for (int i = 0; i < num_samples * len_x; ++i) {
    ASSERT_NEAR(res[i], lhs[i] * rhs[i], 1e-5);
  }
}

TEST_F(TestFunctionPointer, testDivByFunctionPointer) {
  res = InvokeBinaryOperation(p_div, lhs, num_samples, rhs);

  for (int i = 0; i < num_samples * len_x; ++i) {
    ASSERT_NEAR(res[i], lhs[i] / rhs[i], 1e-5);
  }
}
