#include "hip/hip_runtime.h"
// Copyright © 2024 ADR Team
//
// Permission is hereby granted, free of charge, to any person obtaining
// a copy of this software and associated documentation files (the "Software"),
// to deal in the Software without restriction, including without limitation
// the rights to use, copy, modify, merge, publish, distribute, sublicense,
// and/or sell copies of the Software, and to permit persons to whom the
// Software is furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included
// in all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
// EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
// OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
// IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM,
// DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
// TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE
// OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

#include <random>

#include "gtest/gtest.h"
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"

// Kernel - Adding two matrices MatA and MatB

__global__ void MatAdd(double const* lhs, dim3 size, double const* rhs,
                       double* res) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
  auto rows = size.x;
  auto cols = size.y;
  if (i < rows && j < cols) {
    res[i * cols + j] = lhs[i * cols + j] + rhs[i * cols + j];
  }
}

struct TestBasicKernelLaunch : public testing::Test {
  std::random_device dev;
  std::mt19937 rng{dev()};
  std::uniform_real_distribution<> dist{-10.0, 10.0};
  double Generate() { return dist(rng); };
};

TEST_F(TestBasicKernelLaunch, testBasicKernelLaunch) {
  const ulong rows = 100;
  const ulong cols = 100;
  // Matrix addition kernel launch from host code
  const dim3 threads_per_block(16, 16);

  double* d_lhs;
  ASSERT_TRUE(hipMalloc(&d_lhs, rows * cols * sizeof d_lhs) == hipSuccess);

  double* d_rhs;
  ASSERT_TRUE(hipMalloc(&d_rhs, rows * cols * sizeof d_lhs) == hipSuccess);

  auto* h_lhs = new double[rows * cols];
  auto* h_rhs = new double[rows * cols];
  for (int i = 0; i < rows * cols; ++i) {
    h_lhs[i] = Generate();
    h_rhs[i] = Generate();
  }
  ASSERT_TRUE(hipMemcpy(d_lhs, h_lhs, rows * cols * sizeof d_lhs,
                         hipMemcpyHostToDevice) == hipSuccess);
  ASSERT_TRUE(hipMemcpy(d_rhs, h_rhs, rows * cols * sizeof d_rhs,
                         hipMemcpyHostToDevice) == hipSuccess);

  dim3 num_blocks{static_cast<unsigned int>((rows + threads_per_block.x - 1) /
                                            threads_per_block.x),
                  static_cast<unsigned int>((cols + threads_per_block.y - 1) /
                                            threads_per_block.y)};

  double* d_res;
  ASSERT_TRUE(hipMalloc(&d_res, rows * cols * sizeof d_res) == hipSuccess);
  MatAdd<<<num_blocks, threads_per_block>>>(d_lhs, dim3{rows, cols}, d_rhs,
                                            d_res);
  auto* h_res = new double[rows * cols];
  ASSERT_TRUE(hipMemcpy(h_res, d_res, rows * cols * sizeof h_res,
                         hipMemcpyDeviceToHost) == hipSuccess);
  for (int i = 0; i < rows * cols; ++i) {
    ASSERT_NEAR(h_res[i], h_lhs[i] + h_rhs[i], 1e-5);
  }

  ASSERT_TRUE(hipFree(d_lhs) == hipSuccess);
  ASSERT_TRUE(hipFree(d_rhs) == hipSuccess);
  ASSERT_TRUE(hipFree(d_res) == hipSuccess);
  delete[] h_lhs;
  delete[] h_rhs;
  delete[] h_res;
}

TEST_F(TestBasicKernelLaunch, testBasicKernelLaunchWithThrust) {
  const ulong rows = 100;
  const ulong cols = 100;
  // Matrix addition kernel launch from host code
  const dim3 threads_per_block(16, 16);

  thrust::host_vector<double> h_lhs(rows * cols);
  thrust::generate(h_lhs.begin(), h_lhs.end(), [this] { return Generate(); });
  thrust::device_vector<double> d_lhs(h_lhs.cbegin(), h_lhs.cend());

  thrust::host_vector<double> h_rhs(rows * cols);
  thrust::generate(h_rhs.begin(), h_rhs.end(), [this] { return Generate(); });
  thrust::device_vector<double> d_rhs(h_rhs.cbegin(), h_rhs.cend());

  dim3 num_blocks{static_cast<unsigned int>((rows + threads_per_block.x - 1) /
                                            threads_per_block.x),
                  static_cast<unsigned int>((cols + threads_per_block.y - 1) /
                                            threads_per_block.y)};

  thrust::device_vector<double> d_res(rows * cols);
  MatAdd<<<num_blocks, threads_per_block>>>(
      d_lhs.data().get(), dim3{rows, cols}, d_rhs.data().get(),
      d_res.data().get());
  thrust::host_vector<double> h_res(d_res.begin(), d_res.end());

  for (int i = 0; i < rows * cols; ++i) {
    ASSERT_NEAR(h_res[i], h_lhs[i] + h_rhs[i], 1e-5);
  }
}
